#include "monolithic.h"

// prints error number and exits if error is detected
void cu_error_check(hipError_t error) {

    if (error) {
        printf("Error #%d occured. Exiting. \n", error);
        exit(error);
    }

}

// init cuda and check for possible errors/compatiblility issues
void init() {

    printf("Initializing CUDA...\n");
    hipInit(0);

    int dev_count;
    cu_error_check(hipGetDeviceCount(&dev_count));
    
    if (dev_count == 0) {
        printf("There are no devices that support CUDA.\n");
        exit (0);
    }

    // get handle for device 0
    hipDevice_t dev;
    cu_error_check(hipDeviceGet(&dev, 0));

    // get dev 0 properties
    hipDeviceProp_t dev_prop;
    cu_error_check((hipError_t) hipGetDeviceProperties(&dev_prop, dev));
    
    // ensure device arch is volta or higher
    if (dev_prop.major < 7) {
        printf("cudaTensorCoreGemm requires SM 7.0 or higher to use Tensor Cores.  Exiting...\n");
        exit(-1);
    }

    printf("Everything looks good.\n");

}